#include <hip/hip_runtime.h>

__global__ void squareSumKernel(const float* X, float* partialSums, size_t size) {
    extern __shared__ float sdata[];
    
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    sdata[tid] = 0.0f;
    
    if (i < size) {
        float x = X[i];
        sdata[tid] = x * x;
    }
    
    __syncthreads();
    
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        partialSums[blockIdx.x] = sdata[0];
    }
}

__global__ void normalizeKernel(const float* X, float* Y, float norm, size_t size) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < size) {
        Y[i] = (norm > 0.0f) ? X[i] / norm : X[i];
    }
}

extern "C" void solution(const float* X, float* Y, size_t size) {
    const int threadsPerBlock = 256;
    const int blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    
    float* d_partialSums;
    hipMalloc(&d_partialSums, blocks * sizeof(float));
    
    size_t sharedMemSize = threadsPerBlock * sizeof(float);
    
    squareSumKernel<<<blocks, threadsPerBlock, sharedMemSize>>>(X, d_partialSums, size);
    
    float* h_partialSums = new float[blocks];
    hipMemcpy(h_partialSums, d_partialSums, blocks * sizeof(float), hipMemcpyDeviceToHost);
    
    float sum = 0.0f;
    for (int i = 0; i < blocks; i++) {
        sum += h_partialSums[i];
    }
    
    float norm = sqrtf(sum);
    
    delete[] h_partialSums;
    
    normalizeKernel<<<blocks, threadsPerBlock>>>(X, Y, norm, size);
    
    hipFree(d_partialSums);
}